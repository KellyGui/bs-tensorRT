#include "hip/hip_runtime.h"
#include "preprocess.h"
#include "string.h"
#include <opencv2/opencv.hpp>



__global__ void bilinear_kernel(uint8_t* src, int src_width, int src_height, float* dst, int dst_width, int dst_height, uint8_t const_value_st,
    AffineMatrix d2s, int edge)
{
    int position = blockDim.x*blockIdx.x + threadIdx.x;
    if(position>edge) return;

    float m_x1 = d2s.value[0];
    float m_y1 = d2s.value[1];
    float m_z1 = d2s.value[2];
    float m_x2 = d2s.value[3];
    float m_y2 = d2s.value[4];
    float m_z2 = d2s.value[5];

    int dx = position % dst_width;
    int dy = position / dst_width;
    float src_x = m_x1 * dx + m_y1 * dy + m_z1 + 0.5f;
    float src_y = m_x2 * dx + m_y2 * dy + m_z2 + 0.5f;
    float c_r, c_g, c_b;

    if (src_x <= -1 || src_x >= src_width || src_y <= -1 || src_y >= src_height) {
        // out of range
        c_r = const_value_st;
        c_g = const_value_st;
        c_b = const_value_st;
    } else {
        int x1 = floorf(src_x);
        int y1 = floorf(src_y);
        int x2 = x1+1;
        int y2 = y1+1;

        uint8_t* Q11_b = src+y1*3*src_width+3*x1;
        uint8_t* Q11_g = src+y1*3*src_width+x1*3+1;
        uint8_t* Q11_r = src+y1*3*src_width+3*x1+2;

        uint8_t* Q12_b = src+y2*3*src_width+x1*3;
        uint8_t* Q12_g = Q12_b+1;
        uint8_t* Q12_r = Q12_g+1;
        
        uint8_t* Q22_b = src+y2*3*src_width+x2*3;
        uint8_t* Q22_g = Q22_b+1;
        uint8_t* Q22_r = Q22_g+1;

        uint8_t* Q21_b = src+y1*3*src_width+x2*3;
        uint8_t* Q21_g = Q21_b+1;
        uint8_t* Q21_r = Q21_g+1;
    
        float R1_r = (float)(*Q11_r)*(x2-src_x)/(x2-x1)+ (float)(*Q21_r)*(src_x-x1)/(x2-x1);
        float R1_g = (float)(*Q11_g)*(x2-src_x)/(x2-x1)+ (float)(*Q21_g)*(src_x-x1)/(x2-x1);
        float R1_b = (float)(*Q11_b)*(x2-src_x)/(x2-x1)+ (float)(*Q21_b)*(src_x-x1)/(x2-x1);

        float R2_r = (float)(*Q12_r)*(x2-src_x)/(x2-x1)+ (float)(*Q22_r)*(src_x-x1)/(x2-x1);
        float R2_g = (float)(*Q12_g)*(x2-src_x)/(x2-x1)+ (float)(*Q22_g)*(src_x-x1)/(x2-x1);
        float R2_b = (float)(*Q12_b)*(x2-src_x)/(x2-x1)+ (float)(*Q22_b)*(src_x-x1)/(x2-x1);

        c_r = R1_r*(y2-src_y)/(y2-y1)+ R2_r*(src_y-y1)/(y2-y1);
        c_g = R1_g*(y2-src_y)/(y2-y1)+ R2_g*(src_y-y1)/(y2-y1);
        c_b = R1_b*(y2-src_y)/(y2-y1)+ R2_b*(src_y-y1)/(y2-y1);
        
        //normalize
        // mean 103.53, 116.28, 123.675
        // std  57.375, 57.12, 58.395
        c_r = (c_r-103.53)/57.375;
        c_g = (c_g-116.28)/57.12;
        c_b = (c_g- 123.675)/58.395;

        int area = dst_width*dst_height;
        float* pdst_cb = dst + dy * dst_width + dx;
        float* pdst_cg = pdst_cb + area;
        float* pdst_cr = pdst_cg + area;
        *pdst_cr = c_r;
        *pdst_cg = c_g;
        *pdst_cb = c_b;

    }

}

__global__ void warpaffine_kernel( 
    uint8_t* src, int src_line_size, int src_width, 
    int src_height, float* dst, int dst_width, 
    int dst_height, uint8_t const_value_st,
    AffineMatrix d2s, int edge) {
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= edge) return;

    float m_x1 = d2s.value[0];
    float m_y1 = d2s.value[1];
    float m_z1 = d2s.value[2];
    float m_x2 = d2s.value[3];
    float m_y2 = d2s.value[4];
    float m_z2 = d2s.value[5];

    int dx = position % dst_width;
    int dy = position / dst_width;
    float src_x = m_x1 * dx + m_y1 * dy + m_z1 + 0.5f;
    float src_y = m_x2 * dx + m_y2 * dy + m_z2 + 0.5f;
    float c0, c1, c2;

    if (src_x <= -1 || src_x >= src_width || src_y <= -1 || src_y >= src_height) {
        // out of range
        c0 = const_value_st;
        c1 = const_value_st;
        c2 = const_value_st;
    } else {
        int y_low = floorf(src_y);
        int x_low = floorf(src_x);
        int y_high = y_low + 1;
        int x_high = x_low + 1;

        uint8_t const_value[] = {const_value_st, const_value_st, const_value_st};
        float ly = src_y - y_low;
        float lx = src_x - x_low;
        float hy = 1 - ly;
        float hx = 1 - lx;
        float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
        uint8_t* v1 = const_value;
        uint8_t* v2 = const_value;
        uint8_t* v3 = const_value;
        uint8_t* v4 = const_value;

        if (y_low >= 0) {
            if (x_low >= 0)
                v1 = src + y_low * src_line_size + x_low * 3;

            if (x_high < src_width)
                v2 = src + y_low * src_line_size + x_high * 3;
        }

        if (y_high < src_height) {
            if (x_low >= 0)
                v3 = src + y_high * src_line_size + x_low * 3;

            if (x_high < src_width)
                v4 = src + y_high * src_line_size + x_high * 3;
        }

        c0 = w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0];
        c1 = w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1];
        c2 = w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2];
    }

    //bgr to rgb 
    float t = c2;
    c2 = c0;
    c0 = t;

    //normalization
    c0 = c0 / 255.0f;
    c1 = c1 / 255.0f;
    c2 = c2 / 255.0f;

    //rgbrgbrgb to rrrgggbbb
    int area = dst_width * dst_height;
    float* pdst_c0 = dst + dy * dst_width + dx;
    float* pdst_c1 = pdst_c0 + area;
    float* pdst_c2 = pdst_c1 + area;
    *pdst_c0 = c0;
    *pdst_c1 = c1;
    *pdst_c2 = c2;
}


void preprocess_kernel_img(
    uint8_t* src, int src_width, int src_height,
    float* dst, int dst_width, int dst_height,
    hipStream_t stream) {
    AffineMatrix s2d,d2s;
    float scale = std::min(dst_height / (float)src_height, dst_width / (float)src_width);

    s2d.value[0] = scale;
    s2d.value[1] = 0;
    s2d.value[2] = -scale * src_width  * 0.5  + dst_width * 0.5;
    s2d.value[3] = 0;
    s2d.value[4] = scale;
    s2d.value[5] = -scale * src_height * 0.5 + dst_height * 0.5;

    cv::Mat m2x3_s2d(2, 3, CV_32F, s2d.value);
    cv::Mat m2x3_d2s(2, 3, CV_32F, d2s.value);
    cv::invertAffineTransform(m2x3_s2d, m2x3_d2s);

    memcpy(d2s.value, m2x3_d2s.ptr<float>(0), sizeof(d2s.value));  // copy m2x3_d2s to d2s

    int jobs = dst_height * dst_width;
    int threads = 256;
    int blocks = ceil(jobs / (float)threads);
    // bilinear_kernel<<<blocks, threads, 0, stream>>>(
    //     src, src_width,
    //     src_height, dst, dst_width,
    //     dst_height, 128, d2s, jobs);

    warpaffine_kernel<<<blocks, threads, 0, stream>>>(
        src, src_width*3, src_width,
        src_height, dst, dst_width,
        dst_height, 128, d2s, jobs);
}


// int main()
// {
//     hipSetDevice(0);
//     std::String name;
//     cin>>name;
//     cv::Mat img = cv::imread(name);
//     uint8_t* img_host = nullptr;
//     memcpy(img_host,img.data,3*img.cols*img.rows);
//     float* buffer;
//     hipStream_t stream;
//     hipMalloc((void**)&buffer,3*img.cols*img.rows, hipMemcpyHostToDevice);
//     preprocess_kernel_img(img_host, img.cols, img.rows, buffer, 1024, 1024, stream, stream );                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                            )
//     //
// }